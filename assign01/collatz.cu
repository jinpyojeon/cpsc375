
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void testCollatz(long n, long blockSize, long* counterEx) {

	long lowRange = ceil(n * 1.0 / blockSize) * blockIdx.x;
	long highRange = ceil(n * 1.0 / blockSize) * (blockIdx.x + 1);

	long i;
	for (i = lowRange; i < highRange && i <= n; i++) {
		long temp = i;
		int iteration = 0;
		if (temp == 0) continue;
		while (temp != 1) {
			iteration++;
			if (iteration >= 1000) { 
				*counterEx = i; 
				break;
			}
			if (temp % 2 == 0) temp /= 2;
			else temp = (3 * temp) + 1;
		}
	}
}

int main(int argc, char**argv){

	long N, B;
	long* h_counterEx, *d_counterEx;
	if (argc >= 2) {
		N = strtol(argv[1], NULL, 10);
		B = strtol(argv[2], NULL, 10);
	} else {
		return -1;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	h_counterEx = (long*)malloc(sizeof(long));
	*h_counterEx = -1;
	hipMalloc((void**) &d_counterEx, sizeof(long));
	hipMemcpy(d_counterEx, h_counterEx, sizeof(long), hipMemcpyHostToDevice);
	
	testCollatz<<<B,1>>>(N, B, d_counterEx); 

	hipMemcpy(h_counterEx, d_counterEx, sizeof(long), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime = -1;
	hipEventElapsedTime(&elapsedTime,start, stop);
	if (*h_counterEx == -1) {
		printf("Verifying %ld took %f s\n", N, elapsedTime / 1000.0);
	} else {
		printf("Found a counterexample: %ld\n", *h_counterEx);
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
