// Jin Pyo Jeon
// Lab 02
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <math.h>

#define T 1024 // Shared needs to be known at compile time??

__global__ void calculateDot(int N, int* a, int* b, unsigned long* result){
	__shared__ int temp[T];

	int lowRange = ceil(N / (T * 1.0)) * threadIdx.x;
	int highRange = ceil(N / (T * 1.0)) * (threadIdx.x + 1);
	unsigned long sum = 0;

	int i = lowRange;
	for (; i < highRange; i++) {
		sum += a[i] * b[i];
	}

	temp[threadIdx.x] = sum;

	__syncthreads();

	if (0 == threadIdx.x) {
		unsigned long sum = 0;
		for (int i = 0; i < T; i++) {
			sum += temp[i];
		}
		*result = sum;
	}

}

void random_ints(int * arr, size_t size){
	int i = 0;
	for (i = 0; i < size; i++) {
		arr[i] = rand() % 2;
	}
}

int main(int argc, char**argv) {
	unsigned long N;
	if (argc >= 2) {
		N = strtol(argv[1], NULL, 10);
	} else {
		return 0;
	}
	srand(time(NULL));
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	

	int *a, *b;
    unsigned long *c;
	int * d_a, *d_b;
    unsigned long	*d_c;
	int size = N * sizeof(int);

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, sizeof(unsigned long));

	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (unsigned long *)malloc(sizeof(unsigned long));


	random_ints(a, N);
	random_ints(b, N);


	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	calculateDot<<<1, T>>>(N, d_a, d_b, d_c);

	hipMemcpy(c, d_c, sizeof(unsigned long), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("The dot product is %lu with elapsed time of %f s\n", *c, elapsedTime / 1000.0);


	free(a); free(b); free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
